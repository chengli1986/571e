//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

//
// kernel routine
// 

__global__ void add_block(int *a, int *b, int *c)
{
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   c[idx] = a[idx] + b[idx];
}

//
// main code
//

int main(int argc, char **argv)
{
   int *a, *b, *c;
   int *dev_a, *dev_b, *dev_c;
   int size = N * sizeof(int);
   time_t t;

   // initialise card - legacy code

   //cutilDeviceInit(argc, argv);
  
   srand((unsigned) time(&t));
   printf("DEBUG: Size of 'int' type: %lu\n", sizeof(int));

   // allocate device copies of a, b, c
   hipMalloc( (void**)&dev_a, size );
   hipMalloc( (void**)&dev_b, size );
   hipMalloc( (void**)&dev_c, size );

   a = (int*)malloc( size ); 
   b = (int*)malloc( size ); 
   c = (int*)malloc( size );
   
   for (int i=0; i<N; i++)
   {
      a[i] = rand()%N;
      b[i] = rand()%N;
   }
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",0, a[0], 0, b[0]);
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",1, a[1], 1, b[1]);
   
   // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice ); 
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
   
   // launch add() kernel with N parallel blocks
   printf("INFO: Launching CUDA kernel: add_block with blocks=%d, threads=%d...", N/THREADS_PER_BLOCK, THREADS_PER_BLOCK);
   
   add_block<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c );
   
   printf("  Done\n");

   // copy device result back to host copy of c
   hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost );
  
#if 1
   for (int i=0; i<N; i++)
   {
      if (fabs(a[i]+b[i]-c[i]) > 1e-5)
      {
         printf("ERROR: *** FAILED ***\n");
         break;
      } else
      {
         if (i == (N -1))
            printf("INFO: PASS\n");
      }
      //printf("Checking results %d\n", a[i]+b[i]-c[i]);
   }
#endif
#if 1
      printf("DEBUG: a[0]=%d, b[0]=%d, c[0]=%d\n", a[0], b[0], c[0]);
      printf("DEBUG: a[%d]=%d, b[%d]=%d, c[%d]=%d\n", 1, a[1], 1, b[1], 1, c[1]);
      //printf("Checking results %d\n", a[0]+b[0]-c[0]);
#endif

 
   free( a ); 
   free( b ); 
   free( c );

   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );

   hipDeviceReset();

   return 0;
}
