#include "hip/hip_runtime.h"
/*
* Two vector addition using CUDA
* based on prac1b.cu
* Modified by: Aryya Dwisatya W - 13512043
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "cutil_inline.h"


//
// kernel routine
// 

/* Initiatin first vector with value of threads id */
__global__ void my_first_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x;
}

/* Initiatin second vector with value of threads id */
__global__ void my_second_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x; /* udah dengan konstanta jika ingin */
}

/* Adding the value of second vector to the first vector */
__global__ void add_vector(float *x,float *y)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] += y[tid];
}
//
// main code
//

int main(int argc, char **argv)
{
  float *h_x, *d_x,*d_x2;
  int   nblocks, nthreads, nsize, n; 

  // initialise card

  cutilDeviceInit(argc, argv);

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));
  cudaSafeCall(hipMalloc((void **)&d_x, nsize*sizeof(float)));
#if 1
  //h_x2 = (float *)malloc(nsize*sizeof(float));
  cudaSafeCall(hipMalloc((void **)&d_x2, nsize*sizeof(float)));
#endif
  // execute kernel
  
  /* initiating the value of first vector */
  my_first_kernel<<<nblocks,nthreads>>>(d_x);
  cudaCheckMsg("my_first_kernel execution failed\n");
  
  /* initiating the value of second vector */
  my_second_kernel<<<nblocks,nthreads>>>(d_x2);
  cudaCheckMsg("my_first_kernel execution failed\n");
  
  /* Add the second vector to the first */
  add_vector<<<nblocks,nthreads>>>(d_x2, d_x);
  // copy back results and print them out

  /* copy the result to host vector */
  cudaSafeCall( hipMemcpy(h_x,d_x2,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );

	/* print the result */
  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory 

  cudaSafeCall(hipFree(d_x));
  free(h_x);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
