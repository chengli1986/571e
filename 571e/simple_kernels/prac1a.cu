//
// include files
//


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

//////////////////#include <cutil_inline.h>

//
// kernel routine
// 

__global__ void my_first_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x*blockIdx.x;

  x[tid] = (float) threadIdx.x;
}


//
// main code
//

int main(int argc, char **argv)
{
  /* host copy of h_x */
  float *h_x;
  /* device copy of d_x */
  float *d_x;
  int   nblocks, nthreads, nsize; 

  // set number of blocks, and threads per block

  nblocks  = 2;
  nthreads = 8;
  nsize    = nblocks*nthreads ;

  // allocate memory for array
  // reserve memory on HOST
  h_x = (float *)malloc(nsize*sizeof(float));
  // reserve memory on Device
  hipMalloc((void **)&d_x, nsize*sizeof(float));

  // execute kernel

  my_first_kernel<<<nblocks,nthreads>>>(d_x);

  // copy back results and print them out

  hipMemcpy(h_x,d_x,nsize*sizeof(float),hipMemcpyDeviceToHost);

  for (int n=0; n<nsize; n++) 
     printf("INFO: thread count,  thread ID  =  %d  %f \n",n,h_x[n]);

  // free memory 

  hipFree(d_x);
  free(h_x);

  // CUDA exit -- needed to flush printf write buffer

  hipDeviceReset();

  return 0;
}
