
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

__global__
void transpose(float* in, float* out, int width) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x; 
    int ty = blockIdx.y * blockDim.y + threadIdx.y; 
    out[tx * width + ty] = in[ty * width + tx];
}

int main(int args, char** vargs) {
    const int HEIGHT = 1024;
    const int WIDTH = 1024;
    const int SIZE = WIDTH * HEIGHT * sizeof(float); 
    dim3 bDim(16, 16);
    dim3 gDim(WIDTH / bDim.x, HEIGHT / bDim.y);
    float* M = (float*)malloc(SIZE);
    for (int i = 0; i < HEIGHT * WIDTH; i++) { 
        M[i] = i; 
    }
    float* Md = NULL;
    hipMalloc((void**)&Md, SIZE);
    hipMemcpy(Md,M, SIZE, hipMemcpyHostToDevice);
    
    float* Bd = NULL;
    hipMalloc((void**)&Bd, SIZE);
    
    transpose<<<gDim, bDim>>>(Md, Bd, WIDTH); 

    hipMemcpy(M,Bd, SIZE, hipMemcpyDeviceToHost); 

    return 0;
}
