//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

//
// kernel routine
// 

__global__ void dot_product(const int *a, const int *b, int *c)
{
   // each thread in a block sharing the memory, temp
   __shared__ int temp[THREADS_PER_BLOCK];
   int idx = threadIdx.x + blockIdx.x * blockDim.x;
   temp[threadIdx.x] = a[idx] * b[idx];
   
   __syncthreads();

   if (0 == threadIdx.x) {
      int sum = 0;
      /* iterate over only threads in the block */
      for (int i=0; i<THREADS_PER_BLOCK; ++i)
         sum += temp[i];
      /* Tricks: only works for sm_11... read the simpleAtomicIntrinsics sample */
      atomicAdd( c, sum );
   }
}

//
// main code
//

int main(int argc, char **argv)
{
   int *a, *b, *c;
   int *dev_a, *dev_b, *dev_c;
   int size = N * sizeof(int);
   int result = 0; 
   time_t t;

   // initialise card - legacy code

   //cutilDeviceInit(argc, argv);
  
   srand((unsigned) time(&t));
   printf("DEBUG: Size of 'int' type: %lu\n", sizeof(int));
   printf("DEBUG: Total footprint size: %d bytes\n", size);

   // allocate device copies of a, b, c
   hipMalloc( (void**)&dev_a, size );
   hipMalloc( (void**)&dev_b, size );
   hipMalloc( (void**)&dev_c, sizeof(int) );

   a = (int*)malloc( size ); 
   b = (int*)malloc( size ); 
   c = (int*)malloc( sizeof(int) );
   
   for (int i=0; i<N; i++)
   {
#if 0
      a[i] = rand()%N;
      b[i] = rand()%N;
#else
      a[i] = 5;
      b[i] = 5;
#endif
   }
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",0, a[0], 0, b[0]);
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",1, a[1], 1, b[1]);
   
   // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice ); 
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
   // the bug is lacking of this line... sigh
   hipMemcpy( dev_c, c, sizeof(int), hipMemcpyHostToDevice );

   int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
   // launch dot_product() kernel with N parallel blocks
   printf("INFO: Launching CUDA kernel: dot product with blocks=%d, threads=%d...", blocksPerGrid, THREADS_PER_BLOCK);
   
   dot_product<<< blocksPerGrid, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c );
   //dot_product<<< N/THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c );
   
   printf("  Done\n");
   
   printf("DEBUG: c2 is: %d @ %p\n", *c, &c);
   
   // copy device result back to host copy of c
   hipMemcpy( c, dev_c, sizeof(int), hipMemcpyDeviceToHost );
   printf("DEBUG: c3 is: %d @ %p\n", *c, &c);
  
#if 1
   //result = 0;
   for (int i=0; i<N; i++)
   {
      result += a[i] * b[i];
   }
   if (fabs(result - *c) < 1e-5)
      printf("INFO: PASS\n");
   else
      printf("ERROR: *** FAILED *** sum=%d\n", result);
#endif
#if 1
      printf("DEBUG: a[0]=%d, b[0]=%d\n", a[0], b[0]);
      printf("DEBUG: a[%d]=%d, b[%d]=%d, c=%d\n", 1, a[1], 1, b[1], *c);
#endif

   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );

   free( a ); 
   free( b ); 
   free( c );

   hipDeviceReset();

   return 0;
}
