#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 512
#define THREADS_PER_BLOCK 16

//
// kernel routine
// 

__global__ void matrix_add(const int *a, const int *b, int *c)
{
   int col = blockIdx.x * blockDim.x + threadIdx.x;
   int row = blockIdx.y * blockDim.y + threadIdx.y;
   int index = col + row * N;

   if (col < N && row < N)
      c[index] = a[index] + b[index];
}

//
// main code
//

int main(int argc, char **argv)
{

   //int *a, *b, *c;
   int a[N][N], b[N][N], c[N][N];
   int *dev_a, *dev_b, *dev_c;
   int size = N * N * sizeof(int);
   int total;
   
   printf( "DEBUG: Size of 'int' type: %lu\n", sizeof(int) );
   printf( "DEBUG: Total footprint size: %d\n", size );

#if 0   
   // allocate host memory of a, b, c
   a = (int *)malloc( size );
   b = (int *)malloc( size );
   c = (int *)malloc( size );
#endif

   // allocate device copies of a, b, c
   hipMalloc( (void**)&dev_a, size );
   hipMalloc( (void**)&dev_b, size );
   hipMalloc( (void**)&dev_c, size );

   for (int i=0; i<N; i++) {
      for (int j=0; j<N; j++) {
#if 0
      a[i][j] = rand()%N;
      b[i][j] = rand()%N;
      c[i][j] = 0; // init
#else
#if 1
      a[i][j] = 1;
      b[i][j] = 2;
      c[i][j] = 0; // init
#else
      *a = 5; a++;
      *b = 1; b++;
      *c = 0; c++;
#endif
#endif
      }
   }
#if 0
   printf("DEBUG: \n\t");
   for (int i=0; i<N; i++) {
      for (int j=0; j<N; j++) {
         printf("%d ", a[i][j]);
      }
      printf("\n\t");
   } 
   printf("\n\t");

   for (int i=0; i<N; i++) {
      for (int j=0; j<N; j++) {
         printf("%d ", b[i][j]);
      }
      printf("\n\t");
   } 
   printf("\n\t");

   for (int i=0; i<N; i++) {
      for (int j=0; j<N; j++) {
         printf("%d ", c[i][j]);
      }
      printf("\n\t");
   } 
   printf("\n");

#endif
   
   // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice ); 
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );


   dim3 dimBlock ( THREADS_PER_BLOCK, THREADS_PER_BLOCK );
   dim3 dimGrid ( N/THREADS_PER_BLOCK, N/THREADS_PER_BLOCK );
   
   // launch add() kernel with N parallel blocks
   printf("INFO: Launching CUDA kernel: matrix_add with blocks=%d, threads=%d...", 
	N/THREADS_PER_BLOCK, THREADS_PER_BLOCK);
   
   matrix_add<<< dimGrid, dimBlock >>>( dev_a, dev_b, dev_c );
   
   printf("  Done\n");

   // copy device result back to host copy of c
   hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost );
 
#if 0
   printf("\n\t");
   for (int i=0; i<N; i++) {
      for (int j=0; j<N; j++) {
         printf("%d ", c[i][j]);
         total += c[i][j];
      }
      printf("\n\t");
   } 
   printf("\n");
#else
   for (int i=0; i<N; i++) 
      for (int j=0; j<N; j++) 
         total += c[i][j];
#endif

   if ( total == (a[0][0] + b[0][0]) * N * N )
      printf("INFO: PASS: total=%d, c[0][0]=%d\n", total, c[0][0]);
   else
      printf("INFO: FAIL: total=%d, c[0][0]=%d\n", total, c[0][0]);

   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );

   hipDeviceReset();
   
   return 0;
}
