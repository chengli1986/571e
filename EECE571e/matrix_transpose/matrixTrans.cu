
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>


#define N 1024
/* can't change threads number because
 * this example is a 2-D array, which means
 * each blocks will have 16*16 = 256 threads
 * the max number of threads a block can
 * handle 512 threads, so that's why when
 * THREADS_PER_BLOCK is set to 32, it exceeds
 * the number of threads can support for sm_12
 */
#define THREADS_PER_BLOCK 16

__global__
void transpose(float *in, float *out, int width) {
//void transpose(float* in, float* out, int width) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x; 
    int ty = blockIdx.y * blockDim.y + threadIdx.y; 
    out[tx * width + ty] = in[ty * width + tx];
}

int main(int args, char** vargs) {

    hipError_t err = hipSuccess;
    
    const int HEIGHT = N;
    const int WIDTH = N;
    const int SIZE = WIDTH * HEIGHT * sizeof(float); 

    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 blocksPerGrid(N / threadsPerBlock.x, N / threadsPerBlock.y);

    float *M = (float *)malloc(SIZE);

    printf("DEBUG: Size of 'float' type: %lu\n", sizeof(float));
    printf("DEBUG: Footprint total size: %d bytes\n", SIZE);
    

    for (int i = 0; i < HEIGHT * WIDTH; i++) { 
        M[i] = i; 
#if 0
        printf(" %d", i);
        printf(" %f", M[i]);
#endif
    }
#if 0
    printf("DEBUG: \n\t");
    for (int i=0; i<HEIGHTN*WIDTH; i++) {
       printf("%f ", M[i]);
       if ( (i != 0) && (i % N == (N-1)) )
          printf("\n\t");
    } 
    printf("\n");
#else
    printf("DEBUG: \n\t");
    for (int i=0; i<WIDTH; i++) {
       printf("%f ", M[i]);
    } 
    printf("\n");

#endif


    float *Md = NULL;
    err = hipMalloc((void **)&Md, SIZE);

    float *Bd = NULL;
    err = hipMalloc((void **)&Bd, SIZE);

    err = hipMemcpy(Md, M, SIZE, hipMemcpyHostToDevice);
    
    printf("\nINFO: Launching CUDA kernel: transpose with blocks=%d, threads=%d...", 
              N/threadsPerBlock.x, THREADS_PER_BLOCK);
 
    transpose<<<blocksPerGrid, threadsPerBlock>>>(Md, Bd, N); 
    err = hipGetLastError();
 
    printf("  Done\n");
   
    err = hipMemcpy(M, Bd, SIZE, hipMemcpyDeviceToHost); 
#if 0
    printf("DEBUG: \n\t");
    for (int i=0; i<HEIGHT*WIDTH; i++) {
       printf("%f ", M[i]);
       if ( (i != 0) && (i % N == (N-1)) )
          printf("\n\t");
    } 
    printf("\n");
#else
    printf("DEBUG: \n\t");
    for (int i=0; i<HEIGHT*WIDTH; i++) {
       if ( (i == 0) || (i % N == 0) )
          printf("%f ", M[i]);
    } 
    printf("\n");
#endif
    
    printf("DEBUG: visually checking the results\n");

    free(M);
    err = hipFree(Md);
    err = hipFree(Bd);
    err = hipDeviceReset();
    
    err = hipDeviceReset();

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    printf("INFO: Done\n");

    return 0;
}
