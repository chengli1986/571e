
#include <hip/hip_runtime.h>
#include <stdio.h> 
#include <stdlib.h>

#define N 1024
#define THREADS_PER_BLOCK 16

__global__
void transpose(float* in, float* out, int width) {
    int tx = blockIdx.x * blockDim.x + threadIdx.x; 
    int ty = blockIdx.y * blockDim.y + threadIdx.y; 
    out[tx * width + ty] = in[ty * width + tx];
}

int main(int args, char** vargs) {
    const int HEIGHT = N;
    const int WIDTH = N;
    const int SIZE = WIDTH * HEIGHT * sizeof(float); 
    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 blocksPerGrid(N / threadsPerBlock.x, N / threadsPerBlock.y);

    float* M = (float*)malloc(SIZE);

    printf("DEBUG: Size of 'float' type: %lu\n", sizeof(float));
    

    for (int i = 0; i < HEIGHT * WIDTH; i++) { 
        M[i] = i; 
    }

    printf("DEBUG: ");
    for (int i=0; i<10; i++) {
       printf("%f ", M[i]);
    } 
    printf("\n");


    float* Md = NULL;
    hipMalloc((void**)&Md, SIZE);

    hipMemcpy(Md,M, SIZE, hipMemcpyHostToDevice);
    
    float* Bd = NULL;
    hipMalloc((void**)&Bd, SIZE);
   
    printf("INFO: Launching CUDA kernel: transpose with blocks=%d, threads=%d...", 
              16, 1024/16);
 
    /* 64 blocks, 16 threads */ 
    transpose<<<blocksPerGrid, threadsPerBlock>>>(Md, Bd, WIDTH); 
 
    printf("  Done\n");
   
    hipMemcpy(M, Bd, SIZE, hipMemcpyDeviceToHost); 

    printf("DEBUG: ");

    for (int i=0; i<10; i++) {
       printf("%f ", M[i]);
    } 
    printf("\n");

    free(M);
    hipFree(Md);
    hipFree(Bd);
    hipDeviceReset();

    return 0;
}
