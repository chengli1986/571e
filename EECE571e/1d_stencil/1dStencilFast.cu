
#include <hip/hip_runtime.h>
#include <stdio.h>


// We assume that NUM_ELEMENTS is divisible by BLOCK_SIZE
#define RADIUS        3
#define BLOCK_SIZE    256
#define NUM_ELEMENTS  (4096*2)

// CUDA API error checking macro
static void handleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define cudaCheck( err ) (handleError( err, __FILE__, __LINE__ ))

__global__ void stencil_1d(int *in, int *out) 
{
    // __shared__ keyword to declare variables in shared block memory
    __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
    int gindex = threadIdx.x + (blockIdx.x * blockDim.x) + RADIUS;
    int lindex = threadIdx.x + RADIUS;

    temp[lindex] = in[gindex];
    if (threadIdx.x < RADIUS) 
    {
        temp[lindex - RADIUS] = in[gindex - RADIUS];
        temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
    }

    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
        result += temp[lindex + offset];

    // Store the result
    out[gindex-RADIUS] = result;
}

int main()
{
  unsigned int i;

  // vectors stored in the CPU memory - can be used from host code only
  int h_in[NUM_ELEMENTS + 2 * RADIUS], h_out[NUM_ELEMENTS];

  // vectors that will be stored in the device memory - can be dereferenced
  // only in kernel code
  int *d_in, *d_out;

  // Initialize host data
  for( i = 0; i < (NUM_ELEMENTS + 2*RADIUS); ++i )
    h_in[i] = 1; // With a value of 1 and RADIUS of 3, all output values should be 7

  // Allocate space on the device
  // cudaMalloc is equivalent of malloc
  cudaCheck( hipMalloc( &d_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int)) );
  cudaCheck( hipMalloc( &d_out, NUM_ELEMENTS * sizeof(int)) );

  // Copy input data to device
  cudaCheck( hipMemcpy( d_in, h_in, (NUM_ELEMENTS + 2*RADIUS) * sizeof(int), 
  	hipMemcpyHostToDevice) );

  // Call kernels
  stencil_1d<<< (NUM_ELEMENTS + BLOCK_SIZE - 1)/BLOCK_SIZE, BLOCK_SIZE >>> (d_in, d_out);
   
  // Check errors from launching the kernel
  cudaCheck(hipPeekAtLastError());
 
  // Copy results from device memory to host
  cudaCheck( hipMemcpy( h_out, d_out, NUM_ELEMENTS * sizeof(int),
  	hipMemcpyDeviceToHost) );

  // Verify every out value is 7
  for( i = 0; i < NUM_ELEMENTS; ++i )
    if (h_out[i] != 7)
    {
      printf("Element h_out[%d] == %d != 7\n", i, h_out[i]);
      break;
    }

  if (i == NUM_ELEMENTS)
    printf("SUCCESS!\n");

  // Free out memory
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
