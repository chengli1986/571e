//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>

/* when block=1, threads have to be the
 * the maximum based on current kernel
 * implementations  */
#define N 512
#define THREADS_PER_BLOCK 512

//
// kernel routine
// 

//__global__ void dot_product(const int *a, const int *b, int *c, int numElements)
__global__ void dot_product(const int *a, const int *b, int *c)
{
   // each thread in a block sharing the memory, temp
   __shared__ int temp[N];
   temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];
   
   __syncthreads();

   if (threadIdx.x == 0) {
      int sum = 0;
      for (int i=0; i<N; i++)
         sum += temp[i];
      *c = sum;
   }
}

//
// main code
//

int main(int argc, char **argv)
{
   int *a, *b, *c;
   int *dev_a, *dev_b, *dev_c;
   int size = N * sizeof(int);
   int result = 0; 
   time_t t;

   // initialise card - legacy code

   //cutilDeviceInit(argc, argv);
  
   srand((unsigned) time(&t));
   printf("DEBUG: Size of 'int' type: %lu\n", sizeof(int));
   printf("DEBUG: Total footprint size: %d bytes\n", size);

   // allocate device copies of a, b, c
   hipMalloc( (void**)&dev_a, size );
   hipMalloc( (void**)&dev_b, size );
   hipMalloc( (void**)&dev_c, sizeof(int) );

   a = (int*)malloc( size ); 
   b = (int*)malloc( size ); 
   c = (int*)malloc( sizeof(int) );
   
   for (int i=0; i<N; i++)
   {
#if 1
      a[i] = rand()%N;
      b[i] = rand()%N;
#else
      a[i] = 5;
      b[i] = 5;
#endif
   }
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",0, a[0], 0, b[0]);
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",1, a[1], 1, b[1]);
   
   // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice ); 
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );
   
   int threadsPerBlock = THREADS_PER_BLOCK;
   int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

   // launch dot_product() kernel with N parallel threads
   printf("INFO: Launching CUDA kernel: dot product with blocks=%d, threads=%d...", blocksPerGrid, THREADS_PER_BLOCK);
   dot_product<<< blocksPerGrid, THREADS_PER_BLOCK >>>( dev_a, dev_b, dev_c );
   
   printf("  Done\n");

   // copy device result back to host copy of c
   hipMemcpy( c, dev_c, sizeof(int), hipMemcpyDeviceToHost );
  
#if 1
   //result = 0;
   for (int i=0; i<N; i++)
   {
      result += a[i] * b[i];
   }
   if (fabs(result - *c) < 1e-5)
      printf("INFO: PASS\n");
   else
      printf("ERROR: *** FAILED *** sum=%d\n", result);
#endif
#if 1
      printf("DEBUG: a[0]=%d, b[0]=%d\n", a[0], b[0]);
      printf("DEBUG: a[%d]=%d, b[%d]=%d, c=%d\n", 1, a[1], 1, b[1], *c);
      //printf("Checking results %d\n", a[0]+b[0]-c[0]);
#endif

 
   free( a ); 
   free( b ); 
   free( c );

   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );

   hipDeviceReset();

   return 0;
}
