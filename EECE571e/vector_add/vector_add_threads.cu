//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <hip/hip_runtime.h>


#define N 512
//
// kernel routine
// 

__global__ void add_threads(int *a, int *b, int *c)
{
   c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

//
// main code
//

//int main(int argc, char **argv)
int main(void)
{
   int *a, *b, *c;
   int *dev_a, *dev_b, *dev_c;
   int size = N * sizeof(int);
   time_t t;
   
   printf("DEBUG: Size of 'int' type: %lu\n", sizeof(int));
   
   srand((unsigned) time(&t));

   // initialise card

   //cutilDeviceInit(argc, argv);
  
   // allocate device copies of a, b, c
   hipMalloc( (void**)&dev_a, size );
   hipMalloc( (void**)&dev_b, size );
   hipMalloc( (void**)&dev_c, size );

   a = (int*)malloc( size ); 
   b = (int*)malloc( size ); 
   c = (int*)malloc( size );
   
   for (int i=0; i < N; ++i)
   {
#if 0
      a[i] = rand()%N;
      b[i] = rand()%N;
#else
      a[i] = 5;
      b[i] = 5;

#endif

   }
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",0, a[0], 0, b[0]);
   printf("DEBUG: a[%d]=%d, b[%d]=%d\n",N-1, a[N-1], N-1, b[N-1]);
   
   // copy inputs to device
   hipMemcpy( dev_a, a, size, hipMemcpyHostToDevice ); 
   hipMemcpy( dev_b, b, size, hipMemcpyHostToDevice );

   printf("INFO: Launching CUDA kernel: add_block with blocks=%d, threads=%d...", 1, N);

   // launch add() kernel with N parallel blocks
   add_threads<<< 1, N >>>( dev_a, dev_b, dev_c );

   printf("  Done\n");

   // copy device result back to host copy of c
   hipMemcpy( c, dev_c, size, hipMemcpyDeviceToHost );
#if 1
   for (int i=0; i<N; i++)
   {
      if (fabs(a[i]+b[i]-c[i]) > 1e-5)
      {
         printf("ERROR: *** FAILED ***\n");
         break;
      } else
      {
         if (i == (N -1))
            printf("INFO: PASS\n");
      }
      //printf("Checking results %d\n", a[i]+b[i]-c[i]);
   }
#endif
#if 1
      printf("DEBUG: a[0]=%d, b[0]=%d, c[0]=%d\n", a[0], b[0], c[0]);
      printf("DEBUG: a[%d]=%d, b[%d]=%d, c[%d]=%d\n", N-1, a[N-1], N-1, b[N-1], N-1, c[N-1]);
      //printf("Checking results %d\n", a[0]+b[0]-c[0]);
#endif

   free( a ); 
   free( b ); 
   free( c );

   hipFree( dev_a );
   hipFree( dev_b );
   hipFree( dev_c );

   hipDeviceReset();

   return 0;
}
